#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <assert.h>

#include "common.h"

#define L2_CACHE_SIZE           (2 * 1024 * 1024)
#define L2_EVICT_READ_SIZE      (L2_CACHE_SIZE)
#define L2_MIN_STRIDE           (128)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
uint64_t refresh(volatile uint64_t *refresh_vaddr)
{
    uint64_t curindex = 0;
    uint64_t sum = 0;

    while (curindex != (uint64_t)-1) {
        curindex = refresh_vaddr[curindex];
        sum += curindex;
    }
    return sum;
}

__global__
void read_pair(volatile uint64_t *a_v, volatile uint64_t *b_v,
        volatile uint64_t *refresh_v, volatile double *ticks, volatile uint64_t *psum,
        double threshold)
{
    uint64_t curindex;
    uint64_t sum;
    uint64_t count;
    uint64_t mid;
    uint64_t previndex;
    const uint64_t sharednum = MIN_BANK_SIZE/(sizeof(uint64_t));
    __shared__ uint64_t s[sharednum];
    __shared__ uint64_t t[sharednum];
    uint64_t tsum;
    int i;
    double tick;

    for (i = 0; i < MAX_OUTER_LOOP + 1; i++) {

        sum = 0;
        curindex = 0;

        /* Refresh the L2 cache */
        sum += refresh(refresh_v);

        while (curindex != (uint64_t)-1) {
            previndex = curindex;
            mid = clock64();
            sum += b_v[curindex];
            curindex = a_v[curindex];
            s[previndex] = curindex;
            t[previndex] = clock64() - mid;
        }
    
        curindex = 0;
        tsum = 0;
        count = 0;
        while (curindex != (uint64_t)-1) {
            count++;
            tsum += t[curindex];
            //printf("Ticks: %ld, Index: %ld\n", t[curindex], curindex);
            curindex = s[curindex];
        }
       
        /* First run is warmup */
	    if (i == 0)
    	    continue;

        tick = ((double)tsum) / ((double)count);
        if (tick > threshold) {
            /* We don't expect threshold to be crossed on GPU */
            printf("ERROR: Threshold:%f, Ticks:%f, i:%d, count: %ld\n", threshold, tick, i, count);
            i--;
            continue;
        }

        ticks[i - 1] = tick;
        psum[i - 1] = sum;
    }
}

void shuffle(uint64_t *array, size_t n)
{
    size_t i;
    for (i = 0; i < n - 1; i++) 
    {
        size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
        uint64_t t = array[j];
        array[j] = array[i];
        array[i] = t;
    }
}

/* 
 * Initialize the pointer chase for refresh vaddr to hinder an hardware
 * mechanism to predict access pattern
 */
void init_pointer_chase(uint64_t *array, size_t size, size_t offset, int do_shuffle)
{
    uint64_t num_elem = size / offset;
    uint64_t *indexes = (uint64_t *)malloc(size);
    uint64_t curindex;
    uint64_t i;

    assert(offset >= sizeof(uint64_t));
    assert((offset % sizeof(uint64_t)) == 0);
    assert(indexes != NULL);

    for (i = 0; i < num_elem; i++) {
        indexes[i] = i;
    }

    if (do_shuffle)
    	shuffle(indexes, num_elem);

    for (i = 0, curindex = 0; i < num_elem; i++) {
        array[curindex] = indexes[i] * (offset / sizeof(uint64_t));
        curindex = indexes[i] * (offset / sizeof(uint64_t));
    }

    array[curindex] = (uint64_t)-1;

    free(indexes);
}

extern "C" {

void *allocate_gpu_contigous(int contiguous_pages, uintptr_t *phy_start)
{
    size_t size = contiguous_pages * PAGE_SIZE;
    void *gpu_mem;
    int device = -1;

    gpuErrchk(hipMallocManaged(&gpu_mem, size));
   
    /* To print out dmesg */
    gpuErrchk(hipGetDevice(&device));
    gpuErrchk(hipMemPrefetchAsync(gpu_mem, size, device, NULL)); 

    printf("Enter the physical start address:\n");
    scanf("0x%lx", phy_start);
    printf("User input is: 0x%lx\n", *phy_start);

    return gpu_mem;
}

double find_gpu_read_time(void *_a, void *_b, double threshold)
{
    uint64_t *a = (uint64_t *)_a;
    uint64_t *b = (uint64_t *)_b;
    int i;
    double min_ticks, max_ticks, sum_ticks;
    double avg_ticks;
    int device = -1;
    static uint64_t *sum, *refresh_v;
    static double *ticks;
    static int is_initialized = 0;
    static uint64_t *start_v;

    if (is_initialized == 0) {
        gpuErrchk(hipGetDevice(&device));
        gpuErrchk(hipMallocManaged(&refresh_v, L2_EVICT_READ_SIZE));
        gpuErrchk(hipMallocManaged(&ticks, (MAX_OUTER_LOOP) * sizeof(double)));
        gpuErrchk(hipMallocManaged(&sum, (MAX_OUTER_LOOP) * sizeof(uint64_t)));
        
        /* TODO: Why does this whole thing become slow then I make 'true' -> 'false' */
        init_pointer_chase(refresh_v, L2_EVICT_READ_SIZE, sizeof(uint64_t), true);
        start_v = a;

        is_initialized = true;
    }

    init_pointer_chase(a, MIN_BANK_SIZE, L2_MIN_STRIDE, false); 
    gpuErrchk(hipMemPrefetchAsync(start_v, MEM_SIZE, device, NULL));
    gpuErrchk(hipMemPrefetchAsync(refresh_v, L2_EVICT_READ_SIZE, device, NULL));


    gpuErrchk(hipDeviceSynchronize());
    read_pair<<<1,1>>>(a, b, refresh_v, ticks, sum, threshold);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize()); 

    for (i = 0, min_ticks = LONG_MAX, sum_ticks = 0, max_ticks = 0; 
            i < MAX_OUTER_LOOP; i++) {
        double tick = ticks[i];

        assert(tick > 0);
		
        min_ticks = tick < min_ticks ? tick : min_ticks;
        max_ticks = tick > max_ticks ? tick : max_ticks;
        sum_ticks += tick;
    }

    avg_ticks = (sum_ticks * 1.0f) / MAX_OUTER_LOOP;
    dprintf("Avg Ticks: %0.3f,\tMax Ticks: %0.3f,\tMin Ticks: %0.3f\n",
            avg_ticks, max_ticks, min_ticks);
    return avg_ticks;
    
}

} // extern "C"
